#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <sys/mman.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <iostream>
#include <chrono>
#include <cstring>
#include <cassert>
#include <fstream>

struct node {
  node *next;
};

template <typename T>
void swap(T &x, T &y) {
  T t = x;
  x = y; y = t;
}

template <typename T>
void shuffle(std::vector<T> &vec) {
  const size_t len = vec.size();
  for(size_t i = 0; i < len; i++) {
    size_t j = i + (rand() % (len - i));
    swap(vec[i], vec[j]);
  }
}

template <typename T>
size_t partition(T *arr, size_t n) {
  size_t d=0;
  size_t r = rand() % n;
  T p = arr[r];
  arr[r] = arr[n-1];
  arr[n-1] = p;
  
  for(size_t i=0;i<(n-1);i++) {
    if(arr[i] < p) {
      swap(arr[i], arr[d]);
      d++;
    }
  }
  arr[n-1] = arr[d];
  arr[d] = p;
  return d;
}

template <typename T>
void sort(T *arr, size_t len) {
  size_t d;
  if(len <= 16) {
    for(size_t i=1;i<len;i++) {
      size_t j=i;
      while((j > 0) && (arr[j-1] > arr[j])) {
	swap(arr[j-1], arr[j]);
	j--;
      }
    }    
    return;
  }
  d = partition(arr, len);
  sort(arr, d);
  sort(arr+d+1, len-d-1);
}


__global__ void gups(uint64_t *mem, uint32_t *xx, uint32_t lg_n, int64_t iters) {
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
  uint32_t x = xx[idx];
  const uint32_t m = (1U<<lg_n)-1;
  
  while(iters >= 0) {
    mem[x&m] ^= x;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    mem[x&m] ^= x;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    
    iters -= 2;
  }
}

int main(int argc, char *argv[]) {
  static const int warp_size = 32;  
  static const int nthr = warp_size;
  static const uint32_t max_n = 1U<<28;
  static const int64_t iters = 1L<<20;
  
  
  uint64_t *buf = nullptr;
  uint32_t *xx = nullptr;

  node *nodes = nullptr, **nodes_out = nullptr;
  int64_t *cycles = nullptr;
  
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if(deviceProp.kernelExecTimeoutEnabled) {
    std::cout << "Warning : kernel timeout enabled (long runs will fail)\n";
  }
  double freq = deviceProp.clockRate * 1000.0;

  auto ce = hipMallocManaged((void**)&xx, sizeof(uint32_t)*nthr);
  if(ce != hipSuccess) {
    std::cout << hipGetErrorString(ce) << "\n";
    exit(-1);
  }

  
  uint32_t n_keys = 1U<<24;
  std::vector<uint32_t> keys(nthr);
  for(uint32_t i = 0; i < n_keys; i++) {
    keys[i] = i+1;
  }
  shuffle(keys);
  //copy to shared buffer
  for(uint32_t i = 0; i < nthr; i++) {
    xx[i] = keys[i];
  }
  keys.clear();
  assert(hipMallocManaged((void**)&buf, sizeof(uint64_t)*max_n) == hipSuccess);

  for(uint32_t lg_n = 10; (1U<<lg_n) <= max_n; ++lg_n) {

    gups<<<nthr/warp_size, warp_size>>>(buf, xx, lg_n, iters);
    hipDeviceSynchronize();
    auto ce = hipGetLastError();
    if(ce != hipSuccess) {
      std::cerr << "iters = " << iters << " : "
		<< hipGetErrorString(ce) << "\n";
    }
  }
  
#if 0
  assert(hipMallocManaged((void**)&nodes, sizeof(node)*max_keys) == hipSuccess);
  assert(hipMallocManaged((void**)&nodes_out, sizeof(node*)*nthr) == hipSuccess);  
  assert(hipMallocManaged((void**)&cycles, sizeof(int64_t)*nthr) == hipSuccess);

  uint64_t max_iter_step = 1UL<<22;
  
  for(uint64_t n_keys = 1UL<<8; n_keys <= max_keys; n_keys *= 2) {
    node *h = &nodes[keys[0]];
    node *c = h;  
    h->next = h;
    for(uint64_t i = 1; i < n_keys; i++) {
      node *n = &nodes[keys[i]];
      node *t = c->next;
      c->next = n;
      n->next = t;
      c = n;
    }
    uint64_t iters = n_keys*16;
    
    for(int i = 0; i < nthr; i++) {
      //int r = rand() % n_keys;
      node *n = &nodes[keys[11]];
      nodes_out[i] = n;
      cycles[i] = 0;
    }
    
    if(iters < (1UL<<20)) {
      iters = 1UL<<20;
    }

    
    if(iters <= max_iter_step) {
      traverse<<<nthr/warp_size, warp_size>>>(nodes_out, cycles, iters);
      hipDeviceSynchronize();
      auto ce = hipGetLastError();
      if(ce != hipSuccess) {
	std::cerr << "iters = " << iters << " : "
		  << hipGetErrorString(ce) << "\n";
      }
    }
    else {
      for(uint64_t itrs = 0; itrs < iters; itrs += max_iter_step) {
	traverse<<<nthr/warp_size, warp_size>>>(nodes_out, cycles, max_iter_step);
	hipDeviceSynchronize();
	assert(hipGetLastError() == hipSuccess);
      }
      max_iter_step /= 2;
    }
    sort(cycles, nthr);
    double cpl = static_cast<double>(cycles[nthr/2]) / iters;
    double nspl = (cpl/freq) / (1e-9);
    std::cout << sizeof(node)*n_keys << " bytes, GPU cycles per load "
	      << cpl << ", nanosec per load " << nspl << " \n";
    
  }
#endif
  hipFree(nodes);
  hipFree(nodes_out);
  hipFree(cycles);
  return 0;
}
